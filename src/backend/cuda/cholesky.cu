/*******************************************************
 * Copyright (c) 2014, ArrayFire
 * All rights reserved.
 *
 * This file is distributed under 3-clause BSD license.
 * The complete license agreement can be obtained at:
 * http://arrayfire.com/licenses/BSD-3-Clause
 ********************************************************/

#include <common/err_common.hpp>
#include <cholesky.hpp>

#include <platform.hpp>
#include <hipblas.h>
#include <identity.hpp>
#include <iostream>
#include <memory.hpp>
#include <copy.hpp>
#include <triangle.hpp>

#include <math.hpp>
#include <common/err_common.hpp>

namespace cuda
{

//hipsolverStatus_t cusolverDn<>potrf_bufferSize(
//        hipsolverHandle_t handle,
//        hipblasFillMode_t uplo,
//        int n,
//        <> *A,
//        int lda,
//        int *Lwork );
//
//hipsolverStatus_t cusolverDn<>potrf(
//        hipsolverHandle_t handle,
//        hipblasFillMode_t uplo,
//        int n,
//        <> *A, int lda,
//        <> *Workspace, int Lwork,
//        int *devInfo );

template<typename T>
struct potrf_func_def_t
{
    typedef hipsolverStatus_t (*potrf_func_def) (
                              hipsolverHandle_t,
                              hipblasFillMode_t,
                              int,
                              T *, int,
                              T *,
                              int, int *);
};

template<typename T>
struct potrf_buf_func_def_t
{
    typedef hipsolverStatus_t (*potrf_buf_func_def) (
                              hipsolverHandle_t,
                              hipblasFillMode_t,
                              int,
                              T *, int,
                              int *);
};

#define CH_FUNC_DEF( FUNC )                                                     \
template<typename T>                                                            \
typename FUNC##_func_def_t<T>::FUNC##_func_def                                  \
FUNC##_func();                                                                  \
                                                                                \
template<typename T>                                                            \
typename FUNC##_buf_func_def_t<T>::FUNC##_buf_func_def                          \
FUNC##_buf_func();                                                              \

#define CH_FUNC( FUNC, TYPE, PREFIX )                                                           \
template<> typename FUNC##_func_def_t<TYPE>::FUNC##_func_def FUNC##_func<TYPE>()                \
{ return (FUNC##_func_def_t<TYPE>::FUNC##_func_def)&cusolverDn##PREFIX##FUNC; }                 \
                                                                                                \
template<> typename FUNC##_buf_func_def_t<TYPE>::FUNC##_buf_func_def FUNC##_buf_func<TYPE>()    \
{ return (FUNC##_buf_func_def_t<TYPE>::FUNC##_buf_func_def)&cusolverDn##PREFIX##FUNC##_bufferSize; }

CH_FUNC_DEF( potrf )
CH_FUNC(potrf , float  , S)
CH_FUNC(potrf , double , D)
CH_FUNC(potrf , cfloat , C)
CH_FUNC(potrf , cdouble, Z)

template<typename T>
Array<T> cholesky(int *info, const Array<T> &in, const bool is_upper)
{

    Array<T> out = copyArray<T>(in);
    *info = cholesky_inplace(out, is_upper);

    if (is_upper) triangle<T, true , false>(out, out);
    else          triangle<T, false, false>(out, out);

    return out;
}

template<typename T>
int cholesky_inplace(Array<T> &in, const bool is_upper)
{
    dim4 iDims = in.dims();
    int N = iDims[0];

    int lwork = 0;

    hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;
    if(is_upper)
        uplo = HIPBLAS_FILL_MODE_UPPER;

    CUSOLVER_CHECK(potrf_buf_func<T>()(solverDnHandle(),
                                       uplo,
                                       N,
                                       in.get(), in.strides()[1],
                                       &lwork));

    T *workspace = memAlloc<T>(lwork);
    int *d_info = memAlloc<int>(1);

    CUSOLVER_CHECK(potrf_func<T>()(solverDnHandle(),
                                   uplo,
                                   N,
                                   in.get(), in.strides()[1],
                                   workspace, lwork,
                                   d_info));

    memFree(workspace);
    memFree(d_info);

    //FIXME: should return h_info
    return 0;
}

#define INSTANTIATE_CH(T)                                                                   \
    template int cholesky_inplace<T>(Array<T> &in, const bool is_upper);                    \
    template Array<T> cholesky<T>   (int *info, const Array<T> &in, const bool is_upper);   \


INSTANTIATE_CH(float)
INSTANTIATE_CH(cfloat)
INSTANTIATE_CH(double)
INSTANTIATE_CH(cdouble)
}
