/*******************************************************
 * Copyright (c) 2014, ArrayFire
 * All rights reserved.
 *
 * This file is distributed under 3-clause BSD license.
 * The complete license agreement can be obtained at:
 * http://arrayfire.com/licenses/BSD-3-Clause
 ********************************************************/

#include <hip/hip_runtime_api.h>
#include <af/array.h>
#include <af/defines.h>
#include <Array.hpp>
#include <copy.hpp>
#include <kernel/memcopy.hpp>
#include <err_cuda.hpp>

namespace cuda
{

    template<typename T>
    void copyData(T *data, const Array<T> &A)
    {
        // FIXME: Merge this with copyArray
        A.eval();

        Array<T> *out = NULL;
        const T *ptr = NULL;

        if (A.isOwner() || // No offsets, No strides
            A.ndims() == 1 // Simple offset, no strides.
            ) {

            //A.get() gets data with offsets
            ptr = A.get();
        } else {
            //FIXME: Think about implementing eval
            out = copyArray(A);
            ptr = out->get();
        }

        CUDA_CHECK(hipMemcpy(data, ptr,
                              A.elements() * sizeof(T),
                              hipMemcpyDeviceToHost));

        if (out != NULL) delete out;

        return;
    }


    template<typename T>
    Array<T> *copyArray(const Array<T> &A)
    {
        Array<T> *out = createEmptyArray<T>(A.dims());

        if (A.isOwner()) {
            CUDA_CHECK(hipMemcpy(out->get(), A.get(),
                                  A.elements() * sizeof(T),
                                  hipMemcpyDeviceToDevice));
        } else {
            // FIXME: Seems to fail when using Param<T>
            kernel::memcopy(out->get(), out->strides().get(), A.get(), A.dims().get(),
                            A.strides().get(), (uint)A.ndims());
        }
        return out;
    }

    template<typename inType, typename outType>
    void copy(Array<outType> &dst, const Array<inType> &src, outType default_value, double factor)
    {
        ARG_ASSERT(1, (src.dims().ndims() == dst.dims().ndims()));

        kernel::copy(Param<outType>(dst), CParam<inType>(src), src.dims().ndims(), default_value, factor);
    }

#define INSTANTIATE(T)                                                  \
    template void      copyData<T> (T *data, const Array<T> &from);     \
    template Array<T>* copyArray<T>(const Array<T> &A);                 \

    INSTANTIATE(float)
    INSTANTIATE(double)
    INSTANTIATE(cfloat)
    INSTANTIATE(cdouble)
    INSTANTIATE(int)
    INSTANTIATE(uint)
    INSTANTIATE(uchar)
    INSTANTIATE(char)
    INSTANTIATE(intl   )
    INSTANTIATE(uintl  )



#define INSTANTIATE_COPY(SRC_T)                                                       \
    template void copy<SRC_T, float  >(Array<float  > &dst, const Array<SRC_T> &src, float   default_value, double factor); \
    template void copy<SRC_T, double >(Array<double > &dst, const Array<SRC_T> &src, double  default_value, double factor); \
    template void copy<SRC_T, cfloat >(Array<cfloat > &dst, const Array<SRC_T> &src, cfloat  default_value, double factor); \
    template void copy<SRC_T, cdouble>(Array<cdouble> &dst, const Array<SRC_T> &src, cdouble default_value, double factor); \
    template void copy<SRC_T, int    >(Array<int    > &dst, const Array<SRC_T> &src, int     default_value, double factor); \
    template void copy<SRC_T, uint   >(Array<uint   > &dst, const Array<SRC_T> &src, uint    default_value, double factor); \
    template void copy<SRC_T, uchar  >(Array<uchar  > &dst, const Array<SRC_T> &src, uchar   default_value, double factor); \
    template void copy<SRC_T, char   >(Array<char   > &dst, const Array<SRC_T> &src, char    default_value, double factor);

    INSTANTIATE_COPY(float )
    INSTANTIATE_COPY(double)
    INSTANTIATE_COPY(int   )
    INSTANTIATE_COPY(uint  )
    INSTANTIATE_COPY(uchar )
    INSTANTIATE_COPY(char  )

#define INSTANTIATE_COMPLEX_COPY(SRC_T)                                               \
    template void copy<SRC_T, cfloat >(Array<cfloat > &dst, const Array<SRC_T> &src, cfloat  default_value, double factor); \
    template void copy<SRC_T, cdouble>(Array<cdouble> &dst, const Array<SRC_T> &src, cdouble default_value, double factor);

    INSTANTIATE_COMPLEX_COPY(cfloat )
    INSTANTIATE_COMPLEX_COPY(cdouble)

#define INSTANTIATE_UNSUPPORTED_COMPLEX_COPY(cmplxType, T)              \
    template<> void copy(Array<T> &dst, const Array<cfloat> &src,       \
                                    T  default_value, double factor)    \
    {                                                                   \
        TYPE_ERROR(0,(af_dtype) af::dtype_traits<T>::af_type);          \
    }                                                                   \
    template<> void copy(Array<T> &dst, const Array<cdouble> &src,      \
                                        T default_value, double factor) \
    {                                                                   \
        TYPE_ERROR(0,(af_dtype) af::dtype_traits<T>::af_type);          \
    }                                                                   \

    INSTANTIATE_UNSUPPORTED_COMPLEX_COPY(cfloat, double)
    INSTANTIATE_UNSUPPORTED_COMPLEX_COPY(cfloat, float)
    INSTANTIATE_UNSUPPORTED_COMPLEX_COPY(cfloat, int)
    INSTANTIATE_UNSUPPORTED_COMPLEX_COPY(cfloat, uint)
    INSTANTIATE_UNSUPPORTED_COMPLEX_COPY(cfloat, char)
    INSTANTIATE_UNSUPPORTED_COMPLEX_COPY(cfloat, uchar)

}
