/*******************************************************
 * Copyright (c) 2014, ArrayFire
 * All rights reserved.
 *
 * This file is distributed under 3-clause BSD license.
 * The complete license agreement can be obtained at:
 * http://arrayfire.com/licenses/BSD-3-Clause
 ********************************************************/

#include <af/array.h>
#include <af/dim4.hpp>
#include <af/defines.h>
#include <Array.hpp>
#include <identity.hpp>
#include <debug_cuda.hpp>
#include <kernel/identity.hpp>

namespace cuda
{
    template<typename T>
    Array<T> identity(const dim4& dims)
    {
        Array<T> out  = createEmptyArray<T>(dims);
        kernel::identity<T>(out);
        return out;
    }

#define INSTANTIATE_IDENTITY(T)                              \
    template Array<T>  identity<T>    (const af::dim4 &dims);

    INSTANTIATE_IDENTITY(float)
    INSTANTIATE_IDENTITY(double)
    INSTANTIATE_IDENTITY(cfloat)
    INSTANTIATE_IDENTITY(cdouble)
    INSTANTIATE_IDENTITY(int)
    INSTANTIATE_IDENTITY(uint)
    INSTANTIATE_IDENTITY(char)
    INSTANTIATE_IDENTITY(uchar)

}
