#include "hip/hip_runtime.h"
/*******************************************************
 * Copyright (c) 2014, ArrayFire
 * All rights reserved.
 *
 * This file is distributed under 3-clause BSD license.
 * The complete license agreement can be obtained at:
 * http://arrayfire.com/licenses/BSD-3-Clause
 ********************************************************/

#include <complex>
#include <af/dim4.hpp>
#include <af/defines.h>
#include <ArrayInfo.hpp>
#include <Array.hpp>
#include <ireduce.hpp>

#undef _GLIBCXX_USE_INT128
#include <complex>
#include <kernel/ireduce.hpp>
#include <err_cuda.hpp>

using af::dim4;

namespace cuda
{

    template<af_op_t op, typename T>
    void ireduce(Array<T> &out, Array<uint> &loc,
                 const Array<T> &in, const int dim)
    {
        kernel::ireduce<T, op>(out, loc.get(), in, dim);
    }

    template<af_op_t op, typename T>
    T ireduce_all(unsigned *loc, const Array<T> &in)
    {
        return kernel::ireduce_all<T, op>(loc, in);
    }

#define INSTANTIATE(ROp, T)                                             \
    template void ireduce<ROp, T>(Array<T> &out, Array<uint> &loc,      \
                                  const Array<T> &in, const int dim);   \
    template T ireduce_all<ROp, T>(unsigned *loc, const Array<T> &in);  \

    //min
    INSTANTIATE(af_min_t, float  )
    INSTANTIATE(af_min_t, double )
    INSTANTIATE(af_min_t, cfloat )
    INSTANTIATE(af_min_t, cdouble)
    INSTANTIATE(af_min_t, int    )
    INSTANTIATE(af_min_t, uint   )
    INSTANTIATE(af_min_t, char   )
    INSTANTIATE(af_min_t, uchar  )

    //max
    INSTANTIATE(af_max_t, float  )
    INSTANTIATE(af_max_t, double )
    INSTANTIATE(af_max_t, cfloat )
    INSTANTIATE(af_max_t, cdouble)
    INSTANTIATE(af_max_t, int    )
    INSTANTIATE(af_max_t, uint   )
    INSTANTIATE(af_max_t, char   )
    INSTANTIATE(af_max_t, uchar  )
}
