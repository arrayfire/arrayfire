/*******************************************************
 * Copyright (c) 2014, ArrayFire
 * All rights reserved.
 *
 * This file is distributed under 3-clause BSD license.
 * The complete license agreement can be obtained at:
 * http://arrayfire.com/licenses/BSD-3-Clause
 ********************************************************/

#include <af/dim4.hpp>
#include <Array.hpp>
#include <regions.hpp>
#include <kernel/regions.hpp>
#include <err_cuda.hpp>

using af::dim4;

namespace cuda
{

template<typename T>
Array<T>  regions(const Array<char> &in, af_connectivity connectivity)
{
    ARG_ASSERT(2, (connectivity==AF_CONNECTIVITY_4 || connectivity==AF_CONNECTIVITY_8));

    const dim4 dims = in.dims();

    Array<T>  out  = createEmptyArray<T>(dims);

    // Create bindless texture object for the equiv map.
    hipTextureObject_t tex = 0;

    //Use texture objects with compute 3.0 or higher
    if (!std::is_same<T,double>::value) {
        hipResourceDesc resDesc;
        memset(&resDesc, 0, sizeof(resDesc));
        resDesc.resType = hipResourceTypeLinear;
        resDesc.res.linear.devPtr = out.get();

        if (std::is_signed<T>::value)
            resDesc.res.linear.desc.f = hipChannelFormatKindSigned;
        else if (std::is_unsigned<T>::value)
            resDesc.res.linear.desc.f = hipChannelFormatKindUnsigned;
        else
            resDesc.res.linear.desc.f = hipChannelFormatKindFloat;

        resDesc.res.linear.desc.x = sizeof(T)*8; // bits per channel
        resDesc.res.linear.sizeInBytes = dims[0] * dims[1] * sizeof(T);
        hipTextureDesc texDesc;
        memset(&texDesc, 0, sizeof(texDesc));
        texDesc.readMode = hipReadModeElementType;
        CUDA_CHECK(hipCreateTextureObject(&tex, &resDesc, &texDesc, NULL));
    }

    switch(connectivity) {
        case AF_CONNECTIVITY_4:
            ::regions<T, false, 2>(out, in, tex);
            break;
        case AF_CONNECTIVITY_8:
            ::regions<T, true,  2>(out, in, tex);
            break;
    }

    //Iterative procedure(while loop) in kernel::regions
    //does stream synchronization towards loop end. So, it is
    //safe to destroy the texture object
    CUDA_CHECK(hipDestroyTextureObject(tex));

    return out;
}

#define INSTANTIATE(T)\
    template Array<T>  regions<T>(const Array<char> &in, af_connectivity connectivity);

INSTANTIATE(float )
INSTANTIATE(double)
INSTANTIATE(int   )
INSTANTIATE(uint  )
INSTANTIATE(short )
INSTANTIATE(ushort)

}
