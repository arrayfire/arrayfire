/*******************************************************
 * Copyright (c) 2014, ArrayFire
 * All rights reserved.
 *
 * This file is distributed under 3-clause BSD license.
 * The complete license agreement can be obtained at:
 * http://arrayfire.com/licenses/BSD-3-Clause
 ********************************************************/

#include <sparse.hpp>
#include <kernel/sparse.hpp>

#include <stdexcept>
#include <string>

#include <arith.hpp>
#include <cast.hpp>
#include <complex.hpp>
#include <copy.hpp>
#include <common/err_common.hpp>
#include <lookup.hpp>
#include <math.hpp>
#include <platform.hpp>
#include <where.hpp>

namespace cuda
{

using namespace common;
using namespace std;

//hipsparseStatus_t hipsparseZcsr2csc(hipsparseHandle_t handle,
//                                  int m, int n, int nnz,
//                                  const hipDoubleComplex *csrSortedVal,
//                                  const int *csrSortedRowPtr, const int *csrSortedColInd,
//                                  hipDoubleComplex *cscSortedVal,
//                                  int *cscSortedRowInd, int *cscSortedColPtr,
//                                  hipsparseAction_t copyValues,
//                                  hipsparseIndexBase_t idxBase);

template<typename T>
struct csr2csc_func_def_t
{
    typedef hipsparseStatus_t (*csr2csc_func_def)( hipsparseHandle_t,
                                                  int, int, int,
                                                  const T *, const int *, const int *,
                                                  T *, int *, int *,
                                                  hipsparseAction_t,
                                                  hipsparseIndexBase_t);
};

//hipsparseStatus_t hipsparseZdense2csr(hipsparseHandle_t handle,
//                                    int m, int n,
//                                    const hipsparseMatDescr_t descrA,
//                                    const hipDoubleComplex *A, int lda,
//                                    const int *nnzPerRow,
//                                    hipDoubleComplex *csrValA,
//                                    int *csrRowPtrA, int *csrColIndA)
template<typename T>
struct dense2csr_func_def_t
{
    typedef hipsparseStatus_t (*dense2csr_func_def)( hipsparseHandle_t,
                                                    int, int,
                                                    const hipsparseMatDescr_t,
                                                    const T *, int,
                                                    const int *,
                                                    T *,
                                                    int *, int *);
};

//hipsparseStatus_t hipsparseZdense2csc(hipsparseHandle_t handle,
//                                    int m, int n,
//                                    const hipsparseMatDescr_t descrA,
//                                    const hipDoubleComplex *A, int lda,
//                                    const int *nnzPerCol,
//                                    hipDoubleComplex *cscValA,
//                                    int *cscRowIndA, int *cscColPtrA)
template<typename T>
struct dense2csc_func_def_t
{
    typedef hipsparseStatus_t (*dense2csc_func_def)( hipsparseHandle_t,
                                                    int, int,
                                                    const hipsparseMatDescr_t,
                                                    const T *, int,
                                                    const int *,
                                                    T *,
                                                    int *, int *);
};

//hipsparseStatus_t hipsparseZcsr2dense(hipsparseHandle_t handle,
//                                    int m, int n,
//                                    const hipsparseMatDescr_t descrA,
//                                    const hipDoubleComplex *csrValA,
//                                    const int *csrRowPtrA,
//                                    const int *csrColIndA,
//                                    hipDoubleComplex *A, int lda)
template<typename T>
struct csr2dense_func_def_t
{
    typedef hipsparseStatus_t (*csr2dense_func_def)( hipsparseHandle_t,
                                                    int, int,
                                                    const hipsparseMatDescr_t,
                                                    const T *,
                                                    const int *,
                                                    const int *,
                                                    T *, int);
};

//hipsparseStatus_t hipsparseZcsc2dense(hipsparseHandle_t handle,
//                                    int m, int n,
//                                    const hipsparseMatDescr_t descrA,
//                                    const hipDoubleComplex *cscValA,
//                                    const int *cscRowIndA,
//                                    const int *cscColPtrA,
//                                    hipDoubleComplex *A, int lda)
template<typename T>
struct csc2dense_func_def_t
{
    typedef hipsparseStatus_t (*csc2dense_func_def)( hipsparseHandle_t,
                                                    int, int,
                                                    const hipsparseMatDescr_t,
                                                    const T *,
                                                    const int *,
                                                    const int *,
                                                    T *, int);
};

//hipsparseStatus_t hipsparseZnnz(hipsparseHandle_t handle,
//                              hipsparseDirection_t dirA,
//                              int m, int n,
//                              const hipsparseMatDescr_t descrA,
//                              const hipDoubleComplex *A, int lda,
//                              int *nnzPerRowColumn,
//                              int *nnzTotalDevHostPtr)
template<typename T>
struct nnz_func_def_t
{
    typedef hipsparseStatus_t (*nnz_func_def)( hipsparseHandle_t,
                                              hipsparseDirection_t,
                                              int, int,
                                              const hipsparseMatDescr_t,
                                              const T *, int,
                                              int *, int *);
};

//hipsparseStatus_t hipsparseZgthr(hipsparseHandle_t handle,
//                               int nnz,
//                               const hipDoubleComplex *y,
//                               hipDoubleComplex *xVal, const int *xInd,
//                               hipsparseIndexBase_t idxBase)
template<typename T>
struct gthr_func_def_t
{
    typedef hipsparseStatus_t (*gthr_func_def)(hipsparseHandle_t,
                                              int,
                                              const T *,
                                              T*, const int *,
                                              hipsparseIndexBase_t);
};

#define SPARSE_FUNC_DEF( FUNC )                     \
template<typename T>                                \
typename FUNC##_func_def_t<T>::FUNC##_func_def      \
FUNC##_func();

#define SPARSE_FUNC( FUNC, TYPE, PREFIX )                           \
template<> typename FUNC##_func_def_t<TYPE>::FUNC##_func_def        \
FUNC##_func<TYPE>()                                                 \
{ return (FUNC##_func_def_t<TYPE>::FUNC##_func_def)&cusparse##PREFIX##FUNC; }

SPARSE_FUNC_DEF(csr2csc)
SPARSE_FUNC(csr2csc, float,  S)
SPARSE_FUNC(csr2csc, double, D)
SPARSE_FUNC(csr2csc, cfloat, C)
SPARSE_FUNC(csr2csc, cdouble,Z)

SPARSE_FUNC_DEF(dense2csr)
SPARSE_FUNC(dense2csr, float,  S)
SPARSE_FUNC(dense2csr, double, D)
SPARSE_FUNC(dense2csr, cfloat, C)
SPARSE_FUNC(dense2csr, cdouble,Z)

SPARSE_FUNC_DEF(dense2csc)
SPARSE_FUNC(dense2csc, float,  S)
SPARSE_FUNC(dense2csc, double, D)
SPARSE_FUNC(dense2csc, cfloat, C)
SPARSE_FUNC(dense2csc, cdouble,Z)

SPARSE_FUNC_DEF(csr2dense)
SPARSE_FUNC(csr2dense, float,  S)
SPARSE_FUNC(csr2dense, double, D)
SPARSE_FUNC(csr2dense, cfloat, C)
SPARSE_FUNC(csr2dense, cdouble,Z)

SPARSE_FUNC_DEF(csc2dense)
SPARSE_FUNC(csc2dense, float,  S)
SPARSE_FUNC(csc2dense, double, D)
SPARSE_FUNC(csc2dense, cfloat, C)
SPARSE_FUNC(csc2dense, cdouble,Z)

SPARSE_FUNC_DEF(nnz)
SPARSE_FUNC(nnz, float,  S)
SPARSE_FUNC(nnz, double, D)
SPARSE_FUNC(nnz, cfloat, C)
SPARSE_FUNC(nnz, cdouble,Z)

SPARSE_FUNC_DEF(gthr)
SPARSE_FUNC(gthr, float,  S)
SPARSE_FUNC(gthr, double, D)
SPARSE_FUNC(gthr, cfloat, C)
SPARSE_FUNC(gthr, cdouble,Z)

#undef SPARSE_FUNC
#undef SPARSE_FUNC_DEF

// Partial template specialization of sparseConvertDenseToStorage for COO
// However, template specialization is not allowed
template<typename T>
SparseArray<T> sparseConvertDenseToCOO(const Array<T> &in)
{
    Array<uint> nonZeroIdx_ = where<T>(in);
    Array<int> nonZeroIdx = cast<int, uint>(nonZeroIdx_);

    dim_t nNZ = nonZeroIdx.elements();

    Array<int> constDim = createValueArray<int>(dim4(nNZ), in.dims()[0]);

    Array<int> rowIdx = arithOp<int, af_mod_t>(nonZeroIdx, constDim, nonZeroIdx.dims());
    Array<int> colIdx = arithOp<int, af_div_t>(nonZeroIdx, constDim, nonZeroIdx.dims());

    Array<T> values = copyArray<T>(in);
    values.modDims(dim4(values.elements()));
    values = lookup<T, int>(values, nonZeroIdx, 0);

    return createArrayDataSparseArray<T>(in.dims(), values, rowIdx, colIdx, AF_STORAGE_COO);
}

template<typename T, af_storage stype>
SparseArray<T> sparseConvertDenseToStorage(const Array<T> &in)
{
    const int M = in.dims()[0];
    const int N = in.dims()[1];

    // Create Sparse Matrix Descriptor
    hipsparseMatDescr_t descr = 0;
    CUSPARSE_CHECK(hipsparseCreateMatDescr(&descr));
    hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

    int d = -1;
    hipsparseDirection_t dir = HIPSPARSE_DIRECTION_ROW;

    if(stype == AF_STORAGE_CSR) {
        d = M;
        dir = HIPSPARSE_DIRECTION_ROW;
    } else {
        d = N;
        dir = HIPSPARSE_DIRECTION_COLUMN;
    }
    Array<int> nnzPerDir = createEmptyArray<int>(dim4(d));

    int nNZ = -1;
    CUSPARSE_CHECK(nnz_func<T>()(
                        sparseHandle(),
                        dir,
                        M, N,
                        descr,
                        in.get(), in.strides()[1],
                        nnzPerDir.get(), &nNZ));

    Array<int> rowIdx = createEmptyArray<int>(dim4());
    Array<int> colIdx = createEmptyArray<int>(dim4());

    if(stype == AF_STORAGE_CSR) {
        rowIdx = createEmptyArray<int>(dim4(M+1));
        colIdx = createEmptyArray<int>(dim4(nNZ));
    } else {
        rowIdx = createEmptyArray<int>(dim4(nNZ));
        colIdx = createEmptyArray<int>(dim4(N+1));
    }
    Array<T> values = createEmptyArray<T>(dim4(nNZ));

    if(stype == AF_STORAGE_CSR)
        CUSPARSE_CHECK(dense2csr_func<T>()(
                        sparseHandle(),
                        M, N,
                        descr,
                        in.get(), in.strides()[1],
                        nnzPerDir.get(),
                        values.get(), rowIdx.get(), colIdx.get()));
    else
        CUSPARSE_CHECK(dense2csc_func<T>()(
                        sparseHandle(),
                        M, N,
                        descr,
                        in.get(), in.strides()[1],
                        nnzPerDir.get(),
                        values.get(), rowIdx.get(), colIdx.get()));

    // Destory Sparse Matrix Descriptor
    CUSPARSE_CHECK(hipsparseDestroyMatDescr(descr));

    return createArrayDataSparseArray<T>(in.dims(), values, rowIdx, colIdx, stype);
}


// Partial template specialization of sparseConvertStorageToDense for COO
// However, template specialization is not allowed
template<typename T>
Array<T> sparseConvertCOOToDense(const SparseArray<T> &in)
{
    Array<T> dense = createValueArray<T>(in.dims(), scalar<T>(0));

    const Array<T>   values = in.getValues();
    const Array<int> rowIdx = in.getRowIdx();
    const Array<int> colIdx = in.getColIdx();

    kernel::coo2dense<T>(dense, values, rowIdx, colIdx);

    return dense;
}

template<typename T, af_storage stype>
Array<T> sparseConvertStorageToDense(const SparseArray<T> &in)
{
    // Create Sparse Matrix Descriptor
    hipsparseMatDescr_t descr = 0;
    CUSPARSE_CHECK(hipsparseCreateMatDescr(&descr));
    hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

    int M = in.dims()[0];
    int N = in.dims()[1];
    Array<T> dense = createValueArray<T>(in.dims(), scalar<T>(0));
    int d_strides1 = dense.strides()[1];

    if(stype == AF_STORAGE_CSR)
        CUSPARSE_CHECK(csr2dense_func<T>()(
                        sparseHandle(),
                        M, N,
                        descr,
                        in.getValues().get(),
                        in.getRowIdx().get(),
                        in.getColIdx().get(),
                        dense.get(), d_strides1));
    else
        CUSPARSE_CHECK(csc2dense_func<T>()(
                        sparseHandle(),
                        M, N,
                        descr,
                        in.getValues().get(),
                        in.getRowIdx().get(),
                        in.getColIdx().get(),
                        dense.get(), d_strides1));

    // Destory Sparse Matrix Descriptor
    CUSPARSE_CHECK(hipsparseDestroyMatDescr(descr));

    return dense;
}

template<typename T, af_storage dest, af_storage src>
SparseArray<T> sparseConvertStorageToStorage(const SparseArray<T> &in)
{
    using std::shared_ptr;
    in.eval();

    int nNZ = in.getNNZ();
    SparseArray<T> converted = createEmptySparseArray<T>(in.dims(), nNZ, dest);

    if(src == AF_STORAGE_CSR && dest == AF_STORAGE_COO) {
        // Copy colIdx as is
        CUDA_CHECK(hipMemcpyAsync(converted.getColIdx().get(), in.getColIdx().get(),
                                   in.getColIdx().elements() * sizeof(int),
                                   hipMemcpyDeviceToDevice,
                                   cuda::getActiveStream()));

        // cusparse function to expand compressed row into coordinate
        CUSPARSE_CHECK(hipsparseXcsr2coo(
                        sparseHandle(),
                        in.getRowIdx().get(),
                        nNZ, in.dims()[0],
                        converted.getRowIdx().get(),
                        HIPSPARSE_INDEX_BASE_ZERO));

        // Call sort
        size_t pBufferSizeInBytes = 0;
        CUSPARSE_CHECK(hipsparseXcoosort_bufferSizeExt(
                        sparseHandle(),
                        in.dims()[0], in.dims()[1], nNZ,
                        converted.getRowIdx().get(), converted.getColIdx().get(),
                        &pBufferSizeInBytes));
        shared_ptr<char> pBuffer(memAlloc<char>(pBufferSizeInBytes).release(), memFree<char>);

        shared_ptr<int> P(memAlloc<int>(nNZ).release(), memFree<int>);
        CUSPARSE_CHECK(hipsparseCreateIdentityPermutation(sparseHandle(), nNZ, P.get()));

        CUSPARSE_CHECK(hipsparseXcoosortByColumn(
                       sparseHandle(),
                       in.dims()[0], in.dims()[1], nNZ,
                       converted.getRowIdx().get(), converted.getColIdx().get(),
                       P.get(), (void*)pBuffer.get()));

        CUSPARSE_CHECK(gthr_func<T>()(
                       sparseHandle(), nNZ,
                       in.getValues().get(),
                       converted.getValues().get(),
                       P.get(), HIPSPARSE_INDEX_BASE_ZERO));

    } else if (src == AF_STORAGE_COO && dest == AF_STORAGE_CSR) {
        // The cusparse csr sort function is not behaving correctly.
        // So the work around is to convert the COO into row major and then
        // convert it to CSR

        // Deep copy input into temporary COO Row Major
        SparseArray<T> cooT = createArrayDataSparseArray<T>(in.dims(), in.getValues(),
                                                            in.getRowIdx(), in.getColIdx(),
                                                            in.getStorage(), true);

        // Call sort to convert column major to row major
        {
            size_t pBufferSizeInBytes = 0;
            CUSPARSE_CHECK(hipsparseXcoosort_bufferSizeExt(
                            sparseHandle(),
                            cooT.dims()[0], cooT.dims()[1], nNZ,
                            cooT.getRowIdx().get(), cooT.getColIdx().get(),
                            &pBufferSizeInBytes));
            shared_ptr<char> pBuffer(memAlloc<char>(pBufferSizeInBytes).release(), memFree<char>);

            shared_ptr<int> P(memAlloc<int>(nNZ).release(), memFree<int>);
            CUSPARSE_CHECK(hipsparseCreateIdentityPermutation(sparseHandle(), nNZ, P.get()));

            CUSPARSE_CHECK(hipsparseXcoosortByRow(
                           sparseHandle(),
                           cooT.dims()[0], cooT.dims()[1], nNZ,
                           cooT.getRowIdx().get(), cooT.getColIdx().get(),
                           P.get(), (void*)pBuffer.get()));

            CUSPARSE_CHECK(gthr_func<T>()(
                           sparseHandle(), nNZ,
                           in.getValues().get(),
                           cooT.getValues().get(),
                           P.get(), HIPSPARSE_INDEX_BASE_ZERO));

        }

        // Copy values and colIdx as is
        CUDA_CHECK(hipMemcpyAsync(converted.getValues().get(), cooT.getValues().get(),
                                   cooT.getValues().elements() * sizeof(T),
                                   hipMemcpyDeviceToDevice,
                                   cuda::getActiveStream()));
        CUDA_CHECK(hipMemcpyAsync(converted.getColIdx().get(), cooT.getColIdx().get(),
                                   cooT.getColIdx().elements() * sizeof(int),
                                   hipMemcpyDeviceToDevice,
                                   cuda::getActiveStream()));

        // cusparse function to compress row from coordinate
        CUSPARSE_CHECK(hipsparseXcoo2csr(
                        sparseHandle(),
                        cooT.getRowIdx().get(),
                        nNZ, cooT.dims()[0],
                        converted.getRowIdx().get(),
                        HIPSPARSE_INDEX_BASE_ZERO));

        // No need to call CSRSORT

    } else {
        // Should never come here
        AF_ERROR("CUDA Backend invalid conversion combination", AF_ERR_NOT_SUPPORTED);
    }

    return converted;
}

#define INSTANTIATE_TO_STORAGE(T, S)                                                                        \
    template SparseArray<T> sparseConvertStorageToStorage<T, S, AF_STORAGE_CSR>(const SparseArray<T> &in);  \
    template SparseArray<T> sparseConvertStorageToStorage<T, S, AF_STORAGE_CSC>(const SparseArray<T> &in);  \
    template SparseArray<T> sparseConvertStorageToStorage<T, S, AF_STORAGE_COO>(const SparseArray<T> &in);  \

#define INSTANTIATE_COO_SPECIAL(T)                                                                      \
    template<> SparseArray<T> sparseConvertDenseToStorage<T, AF_STORAGE_COO>(const Array<T> &in)        \
    { return sparseConvertDenseToCOO<T>(in); }                                                          \
    template<> Array<T> sparseConvertStorageToDense<T, AF_STORAGE_COO>(const SparseArray<T> &in)        \
    { return sparseConvertCOOToDense<T>(in); }                                                          \

#define INSTANTIATE_SPARSE(T)                                                                           \
    template SparseArray<T> sparseConvertDenseToStorage<T, AF_STORAGE_CSR>(const Array<T> &in);         \
    template SparseArray<T> sparseConvertDenseToStorage<T, AF_STORAGE_CSC>(const Array<T> &in);         \
                                                                                                        \
    template Array<T> sparseConvertStorageToDense<T, AF_STORAGE_CSR>(const SparseArray<T> &in);         \
    template Array<T> sparseConvertStorageToDense<T, AF_STORAGE_CSC>(const SparseArray<T> &in);         \
                                                                                                        \
    INSTANTIATE_COO_SPECIAL(T)                                                                          \
                                                                                                        \
    INSTANTIATE_TO_STORAGE(T, AF_STORAGE_CSR)                                                           \
    INSTANTIATE_TO_STORAGE(T, AF_STORAGE_CSC)                                                           \
    INSTANTIATE_TO_STORAGE(T, AF_STORAGE_COO)                                                           \


INSTANTIATE_SPARSE(float)
INSTANTIATE_SPARSE(double)
INSTANTIATE_SPARSE(cfloat)
INSTANTIATE_SPARSE(cdouble)

#undef INSTANTIATE_TO_STORAGE
#undef INSTANTIATE_COO_SPECIAL
#undef INSTANTIATE_SPARSE

}
