/*******************************************************
 * Copyright (c) 2014, ArrayFire
 * All rights reserved.
 *
 * This file is distributed under 3-clause BSD license.
 * The complete license agreement can be obtained at:
 * http://arrayfire.com/licenses/BSD-3-Clause
 ********************************************************/

#include <Array.hpp>
#include <kernel/transpose_inplace.hpp>
#include <transpose.hpp>
#include <af/dim4.hpp>

using af::dim4;

namespace cuda {

template<typename T>
void transpose_inplace(Array<T> &in, const bool conjugate) {
    if (conjugate) {
        kernel::transpose_inplace<T, true>(in);
    } else {
        kernel::transpose_inplace<T, false>(in);
    }
}

#define INSTANTIATE(T) \
    template void transpose_inplace(Array<T> &in, const bool conjugate);

INSTANTIATE(float)
INSTANTIATE(cfloat)
INSTANTIATE(double)
INSTANTIATE(cdouble)
INSTANTIATE(char)
INSTANTIATE(int)
INSTANTIATE(uint)
INSTANTIATE(uchar)
INSTANTIATE(intl)
INSTANTIATE(uintl)
INSTANTIATE(short)
INSTANTIATE(ushort)

}  // namespace cuda
